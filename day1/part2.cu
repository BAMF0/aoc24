
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include <fstream>
#include <iostream>
#include <numeric>
#include <sstream>
#include <string>

// store number of occurrences of values in a at their value as index in 
// out.
// assume out has size of max(a)
__global__
void count_occurrences(int *out, int *a, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        atomicAdd(&out[a[i]], 1);
}

__global__
void multiply_weighted_occurences(int *out, int *a, int *b, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        out[i] = i * a[i] * b[i];
}

int main()
{
    auto start = std::chrono::high_resolution_clock::now();
    // Get input size
    int n_lines = 0;
    std::string line;
    std::ifstream inputFile("input.txt");
    while (std::getline(inputFile, line))
        ++n_lines;
    int N = n_lines;
    inputFile.close();
    inputFile.clear();

    int *a, *b, *out_a, *out_b, *out_reduce;
    hipMallocManaged(&a, N*sizeof(int));
    hipMallocManaged(&b, N*sizeof(int));
    
    int i = 0;
    inputFile.open("input.txt");
    while (std::getline(inputFile, line))
    {
        std::stringstream ss(line);
        int x, y;
        // no error handling for malformed input
        ss >> x >> y;
        a[i] = x;
        b[i] = y;
        ++i;
    }
    inputFile.close();

    std::sort(a, a + N);
    std::sort(b, b + N);
    int M = 1 + std::max(a[N-1], b[N-1]);
    
    // initialize output
    hipMallocManaged(&out_a, M*sizeof(int));
    hipMallocManaged(&out_b, M*sizeof(int));
    hipMallocManaged(&out_reduce, M*sizeof(int));
    hipMemset(&out_a, 0, M*sizeof(int));
    hipMemset(&out_b, 0, M*sizeof(int));
    hipMemset(&out_reduce, 0, M*sizeof(int));

    int blockSize = 512;
    int numBlocks = (N + blockSize - 1) / blockSize;
    count_occurrences<<<numBlocks, blockSize>>>(out_a, a, N);
    count_occurrences<<<numBlocks, blockSize>>>(out_b, b, N);
    hipDeviceSynchronize();
    multiply_weighted_occurences<<<numBlocks, blockSize>>>(out_reduce, a, b, M);
    hipDeviceSynchronize();


    std::cout << "Answer: " << std::accumulate(out_reduce, out_reduce + M, 0) << std::endl;
    std::cout << "biggest: " << std::max(out_reduce, out_reduce + M) << std::endl;

    hipFree(a);
    hipFree(b);
    hipFree(out_a);
    hipFree(out_b);
    hipFree(out_reduce);

    auto stop = std::chrono::high_resolution_clock::now();
    std::cout << "Took: " << std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count() << "μs" << std::endl;
}
